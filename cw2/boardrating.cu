#include "hip/hip_runtime.h"
#include "boardrating.h"

#include <iostream>

using std::cerr;
using std::endl;

namespace CudaFunctions {

__global__ void board_rating_kernel(const bool* cells_, const int* highest_cells_,
                                    int4* output_) {
  const int x = threadIdx.x;
  const int board_id = blockIdx.x;
  
  // Get pointers to this thread's parts of the buffers
  const bool* cells = cells_ + board_id * BOARD_WIDTH * BOARD_HEIGHT + x;
  int highest_cell = *(highest_cells_ + board_id * BOARD_WIDTH + x);
  int4* const output = output_ + board_id * BOARD_WIDTH + x;
  
  // Shared memory used to store neighbouring highest_cells values
  __shared__ int shared_highest_cells[BOARD_WIDTH];
  
  // Fill our part of the shared memory
  shared_highest_cells[x] = highest_cell;
  __syncthreads();
  
  // Special cases for the well depth
  int well_depth;
  if (x == 0) {
    well_depth = highest_cell - shared_highest_cells[1];
  } else if (x == BOARD_WIDTH - 1) {
    well_depth = highest_cell - shared_highest_cells[BOARD_WIDTH - 1];
  } else {
    well_depth = highest_cell - max(shared_highest_cells[x-1], shared_highest_cells[x+1]);
  }
  
  // Initialise stuff
  int holes = -highest_cell;
  int connected_holes = 0;

  // Look at each cell down the board
  bool last_cell = false;
  for (int y=0 ; y<BOARD_HEIGHT ; ++y) {
    bool cell = *(cells + y*BOARD_WIDTH);

    holes += !cell;
    connected_holes += last_cell && !cell;
    
    last_cell = cell;
  }

  // Fill the output buffer
  *output = make_int4(holes, connected_holes, well_depth, 0);
}

__global__ void reduce_kernel(const int4* column_outputs_, const int* highest_cells_,
                              int4* rating_outputs_) {
  const int board_id = blockIdx.x;
  
  // Get pointers to this thread's parts of the buffers
  const int4* column_outputs = column_outputs_ + board_id * BOARD_WIDTH;
  const int* highest_cells = highest_cells_ + board_id * BOARD_WIDTH;
  int4* rating_output = rating_outputs_ + board_id;
  
  int holes = 0;
  int connected_holes = 0;
  int max_well_depth = 0;
  int pile_height = BOARD_HEIGHT;
  int max_pile_height = 0;
  
  for (int x=0 ; x<BOARD_WIDTH ; ++x) {
    const int4 column_output = column_outputs[x];
    const int highest_cell = highest_cells[x];
    
    holes += column_output.x;
    connected_holes += column_output.y;
    max_well_depth += max(column_output.z, 0);
    
    pile_height = min(pile_height, highest_cell);
    max_pile_height = max(max_pile_height, highest_cell);
  }
  
  *rating_output = make_int4(holes, connected_holes, max_well_depth, pile_height);
}

void handle_error(hipError_t e) {
  if (e == hipSuccess)
    return;

  cerr << "CUDA error: " << hipGetErrorString(e) << endl;
  abort();
}

void board_rating(const bool* cells, const int* highest_cells, int count,
                  int4* board_output) {
  // First pass - compute ratings for each column
  bool* d_cells;
  int* d_highest_cells;
  int4* d_column_output;
  int4* d_board_output;
  
  const int cells_size         = count*BOARD_WIDTH*BOARD_HEIGHT * sizeof(bool);
  const int highest_cells_size = count*BOARD_WIDTH * sizeof(int);
  const int column_output_size = count*BOARD_WIDTH * sizeof(int4);
  const int board_output_size  = count * sizeof(int4);
  
  handle_error(hipMalloc((void**)&d_cells, cells_size));
  handle_error(hipMalloc((void**)&d_highest_cells, highest_cells_size));
  handle_error(hipMalloc((void**)&d_column_output, column_output_size));
  
  handle_error(hipMemcpy(d_cells, cells, cells_size, hipMemcpyHostToDevice));
  handle_error(hipMemcpy(d_highest_cells, highest_cells, highest_cells_size, hipMemcpyHostToDevice));
  
  int threads_per_block = BOARD_WIDTH;
  int blocks = count;
  board_rating_kernel<<<blocks, threads_per_block>>>(d_cells, d_highest_cells, d_column_output);
  
  // Second pass - reduce per-column scores to per-board scores
  handle_error(hipMalloc((void**)&d_board_output, board_output_size));
  
  threads_per_block = 1;
  reduce_kernel<<<blocks, threads_per_block>>>(d_column_output, d_highest_cells, d_board_output);

  hipMemcpy(board_output, d_board_output, board_output_size, hipMemcpyDeviceToHost);
  
  handle_error(hipFree(d_cells));
  handle_error(hipFree(d_highest_cells));
  handle_error(hipFree(d_column_output));
  handle_error(hipFree(d_board_output));
}

} //namespace CudaFunctions
