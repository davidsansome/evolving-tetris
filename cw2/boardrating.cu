#include "hip/hip_runtime.h"
#include "boardrating.h"

#include <iostream>

using std::cerr;
using std::endl;

namespace CudaFunctions {


// Forward declarations
template <int BOARD_WIDTH, int BOARD_HEIGHT>
void _board_rating(const bool* cells, const int* highest_cells, int count,
                   int4* board_output, int* board_output2);

// Specialisations of board_rating
template <> void board_rating<4,4>(const bool* a, const int* b, int c, int4* d, int* e) {
  _board_rating<4,4>(a,b,c,d,e);
}
template <> void board_rating<6,12>(const bool* a, const int* b, int c, int4* d, int* e) {
  _board_rating<6,12>(a,b,c,d,e);
}
template <> void board_rating<7,14>(const bool* a, const int* b, int c, int4* d, int* e) {
  _board_rating<7,14>(a,b,c,d,e);
}
template <> void board_rating<8,16>(const bool* a, const int* b, int c, int4* d, int* e) {
  _board_rating<8,16>(a,b,c,d,e);
}
template <> void board_rating<9,18>(const bool* a, const int* b, int c, int4* d, int* e) {
  _board_rating<9,18>(a,b,c,d,e);
}
template <> void board_rating<10,20>(const bool* a, const int* b, int c, int4* d, int* e) {
  _board_rating<10,20>(a,b,c,d,e);
}
template <> void board_rating<16,16>(const bool* a, const int* b, int c, int4* d, int* e) {
  _board_rating<16,16>(a,b,c,d,e);
}



template <int BOARD_WIDTH, int BOARD_HEIGHT>
__global__ void board_rating_kernel(const bool* cells_, const int* highest_cells_,
                                    int4* output_) {
  const int x = threadIdx.x;
  const int board_id = blockIdx.x;
  
  // Get pointers to this thread's parts of the buffers
  const bool* cells = cells_ + board_id * BOARD_WIDTH * BOARD_HEIGHT + x;
  int highest_cell = *(highest_cells_ + board_id * BOARD_WIDTH + x);
  int4* const output = output_ + board_id * BOARD_WIDTH + x;
  
  // Shared memory used to store neighbouring highest_cells values
  __shared__ int shared_highest_cells[BOARD_WIDTH];
  
  // Fill our part of the shared memory
  shared_highest_cells[x] = highest_cell;
  __syncthreads();
  
  // Special cases for the well depth
  int well_depth;
  if (x == 0) {
    well_depth = highest_cell - shared_highest_cells[1];
  } else if (x == BOARD_WIDTH - 1) {
    well_depth = highest_cell - shared_highest_cells[BOARD_WIDTH - 1];
  } else {
    well_depth = highest_cell - max(shared_highest_cells[x-1], shared_highest_cells[x+1]);
  }
  
  // Initialise stuff
  int holes = -highest_cell;
  int connected_holes = 0;

  // Look at each cell down the board
  bool last_cell = false;
  for (int y=0 ; y<BOARD_HEIGHT ; ++y) {
    bool cell = *(cells + y*BOARD_WIDTH);

    holes += !cell;
    connected_holes += last_cell && !cell;
    
    last_cell = cell;
  }

  // Fill the output buffer
  *output = make_int4(holes, connected_holes, well_depth, 0);
}

template <int BOARD_WIDTH, int BOARD_HEIGHT>
__global__ void reduce_kernel(const int4* column_outputs_, const int* highest_cells_,
                              int4* rating_outputs_, int* rating_outputs2_) {
  const int board_id = blockIdx.x;
  
  // Get pointers to this thread's parts of the buffers
  const int4* column_outputs = column_outputs_ + board_id * BOARD_WIDTH;
  const int* highest_cells = highest_cells_ + board_id * BOARD_WIDTH;
  int4* rating_output = rating_outputs_ + board_id;
  int* rating_output2 = rating_outputs2_ + board_id;
  
  int holes = 0;
  int connected_holes = 0;
  int max_well_depth = 0;
  int pile_height = BOARD_HEIGHT;
  int max_pile_height = 0;
  
  for (int x=0 ; x<BOARD_WIDTH ; ++x) {
    const int4 column_output = column_outputs[x];
    const int highest_cell = highest_cells[x];
    
    holes += column_output.x;
    connected_holes += column_output.y;
    max_well_depth += max(column_output.z, 0);
    
    pile_height = min(pile_height, highest_cell);
    max_pile_height = max(max_pile_height, highest_cell);
  }
  
  *rating_output = make_int4(holes, connected_holes, max_well_depth,
                             BOARD_HEIGHT - pile_height);
  *rating_output2 = max_pile_height - pile_height;
}

void check_cuda_error(const char *msg) {
#ifndef QT_NO_DEBUG
  hipError_t e = hipDeviceSynchronize();
  if(e != hipSuccess) {
      cerr << "CUDA Error " << msg << " : " << hipGetErrorString(e) << endl;
      abort();
  }
  e = hipGetLastError();
  if(e != hipSuccess) {
      cerr << "CUDA Error " << msg << " : " << hipGetErrorString(e) << endl;
      abort();
  }
#endif
}

template <int BOARD_WIDTH, int BOARD_HEIGHT>
void _board_rating(const bool* cells, const int* highest_cells, int count,
                   int4* board_output, int* board_output2) {
  // First pass - compute ratings for each column
  bool* d_cells;
  int* d_highest_cells;
  int4* d_column_output;
  int4* d_board_output;
  int* d_board_output2;
  
  const int cells_size         = count*BOARD_WIDTH*BOARD_HEIGHT * sizeof(bool);
  const int highest_cells_size = count*BOARD_WIDTH * sizeof(int);
  const int column_output_size = count*BOARD_WIDTH * sizeof(int4);
  const int board_output_size  = count * sizeof(int4);
  const int board_output2_size = count * sizeof(int);
  
  hipMalloc((void**)&d_cells, cells_size);
  hipMalloc((void**)&d_highest_cells, highest_cells_size);
  hipMalloc((void**)&d_column_output, column_output_size);
  
  hipMemcpy(d_cells, cells, cells_size, hipMemcpyHostToDevice);
  hipMemcpy(d_highest_cells, highest_cells, highest_cells_size, hipMemcpyHostToDevice);
  
  int threads_per_block = BOARD_WIDTH;
  int blocks = count;
  board_rating_kernel<BOARD_WIDTH, BOARD_HEIGHT>
                     <<<blocks, threads_per_block>>>
                     (d_cells, d_highest_cells, d_column_output);
  check_cuda_error("board_rating_kernel");
  
  hipFree(d_cells);
  
  // Second pass - reduce per-column scores to per-board scores
  hipMalloc((void**)&d_board_output, board_output_size);
  hipMalloc((void**)&d_board_output2, board_output2_size);
  
  threads_per_block = 1;
  reduce_kernel<BOARD_WIDTH, BOARD_HEIGHT>
               <<<blocks, threads_per_block>>>
               (d_column_output, d_highest_cells, d_board_output, d_board_output2);
  check_cuda_error("reduce_kernel");

  hipMemcpy(board_output, d_board_output, board_output_size, hipMemcpyDeviceToHost);
  hipMemcpy(board_output2, d_board_output2, board_output2_size, hipMemcpyDeviceToHost);
  
  hipFree(d_highest_cells);
  hipFree(d_column_output);
  hipFree(d_board_output);
  hipFree(d_board_output2);
}

} //namespace CudaFunctions
